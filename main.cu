#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////
/// PROGRAMACIÓN EN CUDA C/C++
/// Práctica:	BASICO 5 : Sincronización
/// Autor:		Gustavo Gutierrez Martin
/// Fecha:		Octubre 2022
///////////////////////////////////////////////////////////////////////////

/// dependencias ///
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <cmath>

/// constantes ///
#define MB (1<<20) /// MiB = 2^20
#define PI 3.141593F /// numero PI a comparar

/// muestra por consola que no se ha encontrado un dispositivo CUDA
int getErrorDevice();
/// muestra los datos de los dispositivos CUDA encontrados
int getDataDevice(int deviceCount);
/// numero de CUDA cores
int getCudaCores(hipDeviceProp_t deviceProperties);
/// muestra por pantalla las propiedades del dispositivo CUDA
int getDeviceProperties(int deviceId, int cudaCores, hipDeviceProp_t cudaProperties);
/// inicializa el array del host
/// solicita al usuario el número de elementos que se sumaran
int requestNumberOfTerms(int *numberOfItems, int maxThreadsPerBlock);
/// realiza la suma de los arrays en el device
__global__ void reduction(float *dev_datos, float *dev_suma);
/// transferimos los datos del device al host
int dataTransferToHost(float *hst_suma, float *dev_suma);
/// función que muestra por pantalla la salida del programa
int getAppOutput();

int main() {
    int deviceCount, maxThreadsPerBlock;
    int numberOfTerms = 0;
    float pi_calculate, relative_err, abs_err;
    float *hst_suma;
    float *dev_datos,*dev_suma;

    /// buscando dispositivos
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        /// mostramos el error si no se encuentra un dispositivo
        return getErrorDevice();
    } else {
        /// mostramos los datos de los dispositivos CUDA encontrados
        maxThreadsPerBlock = getDataDevice(deviceCount);
    }
    /// solicitamos al usuario la cantidad de elementos
    requestNumberOfTerms(&numberOfTerms, maxThreadsPerBlock);
    /// reserva del espacio de memoria en el host
    hst_suma = (float*)malloc(sizeof(float));
    /// reserva del espacio de memoria en el device
    hipMalloc( (void**)&dev_datos, numberOfTerms * sizeof(float));
    hipMalloc( (void**)&dev_suma, sizeof(float));
    /// imprimimos por pantalla los hilos lanzados
    printf("Lanzamiento de: 1 bloque y %d hilos \n", numberOfTerms);
    /// sumamos los items
    reduction<<< 1, numberOfTerms >>>(dev_datos, dev_suma);
    /// transferimos los datos del device al host
    dataTransferToHost(hst_suma,dev_suma);
    /// calculamos el valor de PI
    pi_calculate = sqrt(6 * hst_suma[0]);
    /// hallamos el error absoluto
    abs_err = pi_calculate - PI;
    /// hallamos el error relativo
    relative_err = (abs_err / PI) * 100;
    printf("> Valor de PI \t\t: %.6f \n", PI);
    printf("> Valor calculado \t: %.6f \n", pi_calculate);
    printf("> Error absoluto \t: %.6f \n", abs_err);
    printf("> Error relativo \t: %.6f%% \n", relative_err);
    /// función que muestra por pantalla la salida del programa
    getAppOutput();
    /// liberamos los recursos del device
    hipFree(dev_datos);
    hipFree(dev_suma);
    return 0;
}

int getErrorDevice() {
    printf("¡No se ha encontrado un dispositivo CUDA!\n");
    printf("<pulsa [INTRO] para finalizar>");
    getchar();
    return 1;
}

int getDataDevice(int deviceCount) {
    printf("Se han encontrado %d dispositivos CUDA:\n", deviceCount);
    int maxThreadsPerBlock = 0;
    for (int deviceID = 0; deviceID < deviceCount; deviceID++) {
        ///obtenemos las propiedades del dispositivo CUDA
        hipDeviceProp_t deviceProp{};
        hipGetDeviceProperties(&deviceProp, deviceID);
        getDeviceProperties(deviceID, getCudaCores(deviceProp), deviceProp);
        maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    }
    return maxThreadsPerBlock;
}

int getCudaCores(hipDeviceProp_t deviceProperties) {
    int cudaCores = 0;
    int major = deviceProperties.major;
    if (major == 1) {
        /// TESLA
        cudaCores = 8;
    } else if (major == 2) {
        /// FERMI
        if (deviceProperties.minor == 0) {
            cudaCores = 32;
        } else {
            cudaCores = 48;
        }
    } else if (major == 3) {
        /// KEPLER
        cudaCores = 192;
    } else if (major == 5) {
        /// MAXWELL
        cudaCores = 128;
    } else if (major == 6 || major == 7 || major == 8) {
        /// PASCAL, VOLTA (7.0), TURING (7.5), AMPERE
        cudaCores = 64;
    } else {
        /// ARQUITECTURA DESCONOCIDA
        cudaCores = 0;
        printf("¡Dispositivo desconocido!\n");
    }
    return cudaCores;
}

int getDeviceProperties(int deviceId, int cudaCores, hipDeviceProp_t cudaProperties) {
    int SM = cudaProperties.multiProcessorCount;
    printf("***************************************************\n");
    printf("DEVICE %d: %s\n", deviceId, cudaProperties.name);
    printf("***************************************************\n");
    printf("- Capacidad de Computo            \t: %d.%d\n", cudaProperties.major, cudaProperties.minor);
    printf("- No. de MultiProcesadores        \t: %d \n", SM);
    printf("- No. de CUDA Cores (%dx%d)       \t: %d \n", cudaCores, SM, cudaCores * SM);
    printf("- Memoria Global (total)          \t: %zu MiB\n", cudaProperties.totalGlobalMem / MB);
    printf("- No. maximo de Hilos (por bloque)\t: %d\n", cudaProperties.maxThreadsPerBlock);
    printf("***************************************************\n");
    return 0;
}

int requestNumberOfTerms(int *numberOfItems, int maxThreadsPerBlock) {
    int status = 0;
    while (status == 0) {
        printf("Introduce el numero de terminos (potencia de 2): \n");
        scanf_s("%d", numberOfItems);
        if (ceil(log2(*numberOfItems)) == floor(log2(*numberOfItems)) && *numberOfItems <= maxThreadsPerBlock ) {
            printf("El numero de elementos elegido es: %d \n", *numberOfItems);
            status = 1;
        } else {
            printf("El numero maximo de terminos no es potencia de 2 o supera el numero maximo de hilos por bloque \n");
        }
    }
    return 0;
}

__global__ void reduction(float *dev_datos, float *dev_suma) {
    /// KERNEL con 1 bloque de N hilos
    unsigned int N = blockDim.x;
    /// indice local de cada hilo
    unsigned int myID = threadIdx.x;
    /// rellenamos el vector de datos
    unsigned int term = myID + 1;
    dev_datos[myID] = (float)(1.0 / (term * term));
    /// sincronizamos para evitar riesgos de tipo RAW
    __syncthreads();
    /// ******************
    /// REDUCCION PARALELA
    /// ******************
    int salto = N / 2;
    /// realizamos log2(N) iteraciones
    while (salto > 0) {
        /// en cada paso solo trabajan la mitad de los hilos
        if (myID < salto) {
            dev_datos[myID] = dev_datos[myID] + dev_datos[myID + salto];
        }
        /// sincronizamos los hilos evitar riesgos de tipo RAW
        __syncthreads();
        salto = salto / 2;
    }
    /// ******************
    /// Solo el hilo no.'0' escribe el resultado final:
    /// evitamos los riesgos estructurales por el acceso a la memoria
    if (myID == 0) {
        *dev_suma = dev_datos[0];
    }
}

int dataTransferToHost(float *hst_suma, float *dev_suma) {
    /// transfiere datos de dev_vector2 a hst_vector2
    hipMemcpy(hst_suma, dev_suma, sizeof(int), hipMemcpyDeviceToHost);
    return 0;
}

int getAppOutput() {
    /// salida del programa
    time_t fecha;
    time(&fecha);
    printf("***************************************************\n");
    printf("Programa ejecutado el: %s", ctime(&fecha));
    printf("***************************************************\n");
    /// capturamos un INTRO para que no se cierre la consola de MSVS
    printf("<pulsa [INTRO] para finalizar>");
    getchar();
    return 0;
}


